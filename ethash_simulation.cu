#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;
#define group_size 8
#define dag_size 1024*1024*1024

#if (__CUDACC_VER_MAJOR__ > 8)
#define SHFL(x, y, z) __shfl_sync(0xFFFFFFFF, (x), (y), (z))
#else
#define SHFL(x, y, z) __shfl((x), (y), (z))
#endif

#define FNV_PRIME 0x01000193
#define fnv(x, y) ((x)*FNV_PRIME ^ (y))

__global__ void test_dag_device(int *dag, int nonce, int *result_device) {
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int gdx = gridDim.x;
    int gdy = gridDim.y;
    int thread_id = bdx * bix + tix;

    int start_nonce[8];
    int end_nonce[8];

    for(int i=0;i<=7;i++){
        start_nonce[i]= nonce + i + thread_id;
    }

    const int thread_id_ingroup = threadIdx.x & (group_size - 1);
    const int mix_idx = thread_id_ingroup & 3;
    

    for(int i=0;i<=group_size-1;i++){

        int mix;
        int init;
        int offset;
        int shuffle[8];

        for(int j=0;j<=7;j++){
            shuffle[j] = SHFL(start_nonce[j], i, group_size);
        }

        switch(mix_idx){
            case 0:
                mix = shuffle[0] + shuffle[1];
                break;
            case 1:
                mix = shuffle[2] + shuffle[3];
                break;
            case 2:
                mix = shuffle[4] + shuffle[5];
                break;
            case 3:
                mix = shuffle[6] + shuffle[7];
                break;
        }

        init = SHFL(shuffle[0], 0, group_size);

        for(int k =0;k<=63;k++){
            offset = fnv(init ^ k, mix)%dag_size;
            offset = SHFL(offset, k%8, group_size);
            mix = fnv(mix, dag[offset]&thread_id_ingroup);
        }

        int shuffle2[8];
        int thread_mix = mix;
        shuffle2[0] = SHFL(thread_mix, 0, group_size);
        shuffle2[1] = SHFL(thread_mix, 1, group_size);
        shuffle2[2] = SHFL(thread_mix, 2, group_size);
        shuffle2[3] = SHFL(thread_mix, 3, group_size);
        shuffle2[4] = SHFL(thread_mix, 4, group_size);
        shuffle2[5] = SHFL(thread_mix, 5, group_size);
        shuffle2[6] = SHFL(thread_mix, 6, group_size);
        shuffle2[7] = SHFL(thread_mix, 7, group_size);

        if(i==thread_id_ingroup){
            end_nonce[0] = shuffle2[0];
            end_nonce[1] = shuffle2[1];
            end_nonce[2] = shuffle2[2];
            end_nonce[3] = shuffle2[3];
            end_nonce[4] = shuffle2[4];
            end_nonce[5] = shuffle2[5];
            end_nonce[6] = shuffle2[6];
            end_nonce[7] = shuffle2[7];
        }
    }
    int sum=0;
    for(int i=0;i<=7;i++){
        sum = sum+end_nonce[i];
    }
    result_device[thread_id%1024] = sum;
}

int main()
{
    double size = dag_size;
    int *dag_host;
    int *dag_device;
    int *result_host;
    int *result_device;

    cout<<"start"<<endl;
    int nonce = 1010;
    dag_host = (int*) malloc(sizeof(int) * size);

    srand(0);

    for(int i=0;i<=size-1;i++){
        if((int)rand()/RAND_MAX > 0.5){
            dag_host[i]=-1;
        }
        else{
            dag_host[i]=1;
        }
    }

    hipMalloc((void**)&dag_device,sizeof(int) *size);
    hipMalloc((void**)&result_device,sizeof(int) *1024);

    hipMemcpy(dag_device,dag_host,sizeof(int) *size,hipMemcpyHostToDevice);

    dim3 gridsize(8192,1,1);
    dim3 blocksize(128,1,1);

    cout<<"Dag generate finished!"<<endl;

    int count = 0;
    while(1){
        test_dag_device<<<gridsize,blocksize>>>(dag_device, nonce, result_device);
        count = count + 1;

        if(count % 10000 == 0)
            cout<<"Hash finished "<<count<<" times!"<<endl;
    }

    // cudaMemcpy(result_host, result_device,sizeof(int) *size,cudaMemcpyDeviceToHost);

    // cudaFree(dag_device);
    // cudaFree(result_device);



}