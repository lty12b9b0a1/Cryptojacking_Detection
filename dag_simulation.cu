#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void test_dag_device(float *dag, float nonce, float *result_device) {
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int gdx = gridDim.x;
    int gdy = gridDim.y;
    int thread_id = bdx * bix + tix;
    float start_nonce = nonce + float(thread_id);

    for(int i =0;i<=63;i++){
        start_nonce = start_nonce + dag[thread_id];
    }

    result_device[thread_id%1024] = start_nonce;
}

int main()
{
    double size = 1024*1024*1024;
    float *dag_host;
    float *dag_device;
    float *result_host;
    float *result_device;

    cout<<"start"<<endl;
    float nonce = 1010;
    dag_host = (float*) malloc(sizeof(float) * size);
    srand(0);
    for(int i=0;i<=size-1;i++){
        if((float)rand()/RAND_MAX > 0.5){
            dag_host[i]=-1;
        }
        else{
            dag_host[i]=1;
        }
    }

    hipMalloc((void**)&dag_device,sizeof(float) *size);
    hipMalloc((void**)&result_device,sizeof(float) *1024);

    hipMemcpy(dag_device,dag_host,sizeof(float) *size,hipMemcpyHostToDevice);


    dim3 gridsize(8192,1,1);
    dim3 blocksize(128,1,1);

    while(1){
        test_dag_device<<<gridsize,blocksize>>>(dag_device, nonce, result_device);
    }

    hipMemcpy(result_host, result_device,sizeof(float) *size,hipMemcpyDeviceToHost);

    hipFree(dag_device);
    hipFree(result_device);



}